#include "hip/hip_runtime.h"
#include <iostream>
#include<hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include "common.h"
#include <>

//Method which takes each precalculated CPU board setting as the input
//Returns the solution till the boeard size
__device__ uint32_t solveGPURecursion(const int N, const int depth = 0, const uint32_t left_neg = 0, const uint32_t mid = 0, 
	const uint32_t right_pos = 0) {
	if (depth == N) {
		return 1;
	}
	uint32_t sum = 0;
	for (uint32_t pos = (((uint32_t)1 << N) - 1) & ~(left_neg | mid | right_pos); pos; pos &= pos - 1) {
		uint32_t bit = pos & -pos;
		sum += solveGPURecursion(N, depth + 1, (left_neg|bit) << 1, mid | bit, (right_pos|bit) >> 1);
	}
	return sum;
}

//Kernel method where each thread index holds the precalcuated board arrangement upto a certain depth
__global__ void NQueenKernel(const int N, const int depth, const uint32_t* const left_vec,const uint32_t* const mid_vec,const uint32_t* const right_vec,
	uint32_t* const result_vec,
	const size_t size) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < size) {
		result_vec[tid] = solveGPURecursion(N, depth, left_vec[tid], mid_vec[tid], right_vec[tid]);
	}
}

//Class declared to precalculate board setting through CPU and load the blocking diagonals and columns 
//till depth using vectors (dynamic arrays).
class NQueenCPU {
public:
	std::vector<uint32_t> left_vec, mid_vec, right_vec;
	void precalculate(int N,int M, uint32_t ex1, uint32_t ex2, int depth = 0, uint32_t left = 0, uint32_t mid = 0, uint32_t right = 0) {
		if (depth == M) {
			left_vec.push_back(left);
			mid_vec.push_back(mid);
			right_vec.push_back(right);
			return;
		}
		for (uint32_t pos = (((uint32_t)1 << N) - 1) & ~(left | mid | right | ex1); pos; pos &= pos - 1) {
			uint32_t bit = pos & -pos;
			precalculate(N, M, ex2, 0, depth + 1, (left | bit) << 1, mid | bit, (right | bit) >> 1);
			ex2 = 0;
		}
	}
};

//Helper method to allocate and copy memory for the kernel launch
uint64_t QueenGPU(const int N, const int depth) {
	NQueenCPU nqe;
	uint32_t excl = (1 << ((N / 2) ^ 0)) - 1;
	//Calculate board solution upto depth
	nqe.precalculate(N, depth,excl, N % 2 ? excl : 0);
	//Size of precalculated board combinations
	const size_t length = nqe.left_vec.size();
	//Vectors storing the blocking diagonals and columns
	uint32_t* left_d_vec;
	uint32_t* mid_d_vec;
	uint32_t* right_d_vec;
	uint32_t sum = 0;
	//Allocate memory 
	hipMalloc((void**)&left_d_vec, sizeof(uint32_t) * length);
	hipMalloc((void**)&mid_d_vec, sizeof(uint32_t) * length);
	hipMalloc((void**)&right_d_vec, sizeof(uint32_t) * length);
	//Variable to store the solution count
	uint32_t* result_device;
	std::vector<uint32_t> result(length);
	hipMalloc((void**)&result_device, sizeof(uint32_t) * length);
	hipMemcpy(left_d_vec, nqe.left_vec.data(), sizeof(uint32_t) * length, hipMemcpyHostToDevice);
	hipMemcpy(mid_d_vec, nqe.mid_vec.data(), sizeof(uint32_t) * length, hipMemcpyHostToDevice);
	hipMemcpy(right_d_vec, nqe.right_vec.data(), sizeof(uint32_t) * length, hipMemcpyHostToDevice);
	//No of threads per block
	const int threadsPerBlock = 16;
	//No of blocks depends on length (No of precalculated combinations) 
	const int noBlocks = (length + threadsPerBlock - 1) / threadsPerBlock;
	//Invoke the kernel
	NQueenKernel << <noBlocks, threadsPerBlock >> > (N, depth, left_d_vec, mid_d_vec, right_d_vec, result_device, length);
	hipMemcpy(result.data(), result_device, sizeof(uint32_t) * length, hipMemcpyDeviceToHost);
	//Iterate through each resut vector
	for (size_t i = 0; i < length; ++i) 
		sum += result[i];
	//Free the memory
	hipFree(mid_d_vec);
	hipFree(left_d_vec);
	hipFree(result_device);
	hipFree(right_d_vec);
	return sum;
}
